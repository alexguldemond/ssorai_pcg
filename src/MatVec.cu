#include <iostream>
#include <iomanip>
#include <cstdio>
#include <cassert>
#include "DenseVector.hpp"
#include "SparseMatrix.hpp"
#include "DevicePtr.cuh"
#include "Parallel.cuh"

int main () {
    const int dim = 2123;
    DenseVector<float> vec = DenseVector<float>::constant(dim, 1);
    SparseMatrix<float> mat = SparseMatrix<float>::triDiagonal(dim, 1, 2, 3);
    
    DevicePtr<float> deviceEntries(&mat.entries[0], mat.nonZeroEntries());
    DevicePtr<int> deviceCols(&mat.cols[0], mat.nonZeroEntries());
    DevicePtr<int> deviceRowPtrs(&mat.rowPtrs[0], dim + 1);

    DevicePtr<float> deviceVec(vec.data(), dim);

    
    DevicePtr<float> deviceResult(dim);

    int blockSize = (mat.nonZeroEntries()/1024) + 1;
    kernel::sparseMatrixVectorProduct<<<blockSize,
	1024,
	mat.nonZeroEntries() *sizeof(float)>>>(deviceEntries.raw(),
					       deviceCols.raw(),
					       deviceRowPtrs.raw(),
					       deviceVec.raw(),
					       dim,
					       deviceResult.raw());
    checkCuda(hipPeekAtLastError());
    
    float result[dim];
    deviceResult.copyToHost(result);
    std::cout << "NNZ: " << mat.nonZeroEntries() << "\n";

    if (result[0] != 5) {
	std::cout << "result[0] is not 5: " << result[0] << "\n";
	return 1;
    }
    for(int i = 1; i < dim - 1; i++) {
	if (result[i] != 6) {
	    std::cout << "result[" << i << "] is not 6: " << result[i] << "\n";
	    //return 1;
	}
    }
    if (result[dim - 1] != 3) {
	std::cout << "result[last] is not 3: " << result[dim - 1] << "\n";
	return 1;
    }

    return 0;
}